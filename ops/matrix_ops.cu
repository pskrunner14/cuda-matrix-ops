#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "utils/devices.cu"
#include "utils/utils.cpp"

#define BLOCK_SIZE 256

// Build using: nvcc -Xcompiler -fPIC -shared -o lib/cuda_mat_mul.so matmul.cu

__global__ void matSum(float *a, float *b, float *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        c[i] = a[i] + b[i];
}

__global__ void matMul(float *a, float *b, float *c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;

    if (row < m && col < k) {
        for (int i = 0; i < n; i++)
            sum += a[row * n + i] * b[i * n + col];
        c[row * k + col] = sum;
    }
}

extern "C" {

    void cuda_mat_sum(float *a, float *b, float *c, int n) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, n * sizeof(float));
        hipMallocManaged(&d_b, n * sizeof(float));
        hipMallocManaged(&d_c, n * sizeof(float));

        hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

        int blockSize = 256;
        int numBlocks = (n + blockSize - 1) / blockSize;

        matSum<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    void cuda_mat_mul(float *a, float *b, float *c, int m, int n, int k) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_b, (n * k) * sizeof(float));
        hipMallocManaged(&d_c, (m * k) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, (n * k) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * k) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}