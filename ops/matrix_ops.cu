#include "hip/hip_runtime.h"
/**
 *  CUDA PARALLEL PROGRAMMING: matrix_ops.cu
 *  Purpose: Matrix Operations using CUDA C/C++
 *  @author Prabhsimran Singh
 *  @version 1.0 17/09/18
 *
 *  Build using: nvcc -Xcompiler -fPIC -shared -o lib/cuda_mat_ops.so matrix_ops.cu
 */

#include <iostream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "utils/devices.cu"
#include "utils/utils.cpp"

#define BLOCK_SIZE 256

/**
 * Calculates element-wise sum of two matrices (using parallel threads on CUDA capable device)
 *
 * @param a the float pointer to first input array
 * @param b the float pointer to second input array
 * @param c the float pointer to output array
 * @param m the no. of rows in the arrays
 * @param n the no. of cols in the arrays
 * @return void
 */
__global__ void matSum(float *a, float *b, float *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
        c[row * n + col] = a[row * n + col] + b[row * n + col];
}

/**
 * Calculates element-wise product of two matrices (using parallel threads on CUDA capable device)
 *
 * @param a the float pointer to first input array
 * @param b the float pointer to second input array
 * @param c the float pointer to output array
 * @param m the no. of rows in the arrays
 * @param n the no. of cols in the arrays
 * @return void
 */
 __global__ void matProd(float *a, float *b, float *c, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < n)
        c[row * n + col] = a[row * n + col] * b[row * n + col];
}

/**
 * Calculates dot-product of two matrices (using parallel threads on CUDA capable device)
 *
 * @param a the float pointer to first input array
 * @param b the float pointer to second input array
 * @param c the float pointer to output array
 * @param m the no. rows in a(m x n) and c(m x k)
 * @param n the no. cols in a(m x n) and rows in b(n x k)
 * @param k the no. cols in b(n x k) and c(m x k)
 * @return void
 */
__global__ void matMul(float *a, float *b, float *c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;

    if (row < m && col < k) {
        for (int i = 0; i < n; i++)
            sum += a[row * n + i] * b[i * n + col];
        c[row * k + col] = sum;
    }
}

extern "C" {

    void cuda_mat_sum(float *a, float *b, float *c, int m, int n) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_b, (m * n) * sizeof(float));
        hipMallocManaged(&d_c, (m * n) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, (m * n) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matSum<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * n) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    void cuda_mat_prod(float *a, float *b, float *c, int m, int n) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_b, (m * n) * sizeof(float));
        hipMallocManaged(&d_c, (m * n) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, (m * n) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matProd<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * n) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }

    void cuda_mat_mul(float *a, float *b, float *c, int m, int n, int k) {
        float *d_a, *d_b, *d_c;

        hipMallocManaged(&d_a, (m * n) * sizeof(float));
        hipMallocManaged(&d_b, (n * k) * sizeof(float));
        hipMallocManaged(&d_c, (m * k) * sizeof(float));

        hipMemcpy(d_a, a, (m * n) * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, (n * k) * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = m / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, (m * k) * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}