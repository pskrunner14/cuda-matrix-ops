#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "utils/devices.cu"

#define BLOCK_SIZE 16

void printMatrix(float* matrix, int N) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            std::cout << matrix[i * N + j] << " ";
        std::cout << std::endl;
    }
}

__global__ void matMul(float* A, float* B, float* C, int m, int n, int k) {

    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;

    // each thread computes one element of the block sub-matrix
    if (ROW < m && COL < k) {
        for (int i = 0; i < n; i++)
            sum += A[ROW * n + i] * B[i * n + COL];
        C[ROW * k + COL] = sum;
    }
}

// Build using: nvcc -Xcompiler -fPIC -shared -o lib/cuda_mat_mul.so matmul.cu

extern "C" {
    void cuda_mat_mul(float *a, float *b, float *c, int N) {
        
        float *d_a, *d_b, *d_c;
        int size = N * N;

        hipMalloc((void **)&d_a, size * sizeof(float));
        hipMalloc((void **)&d_b, size * sizeof(float));
        hipMalloc((void **)&d_c, size * sizeof(float));

        hipMemcpy(d_a, a, size * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size * sizeof(float), hipMemcpyHostToDevice);

        unsigned int grid_rows = sqrt(BLOCK_SIZE);
        unsigned int grid_cols = N / grid_rows;

        dim3 dimGrid(grid_cols, grid_cols, 1);
        dim3 dimBlock(grid_rows, grid_rows, 1);

        matMul<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N, N, N);
        hipDeviceSynchronize();

        hipMemcpy(c, d_c, size * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
}

// int main() {
//     getCudaDeviceInfo();

//     // Perform matrix multiplication C = A*B
//     // where A, B and C are NxN matrices
//     int N = 16;
//     int SIZE = N * N;

//     // Allocate memory on the host
//     float *A, *B, *C;

//     hipMallocManaged(&A, SIZE * sizeof(float));
//     hipMallocManaged(&B, SIZE * sizeof(float));
//     hipMallocManaged(&C, SIZE * sizeof(float));

//     // Initialize matrices on the host
//     for (int i = 0; i < N; i++) {
//         for (int j = 0; j < N; j++) {
//             A[i * N + j] = 2.0f;
//             B[i * N + j] = 3.0f;
//         }
//     }

//     unsigned int grid_rows = sqrt(BLOCK_SIZE);
//     unsigned int grid_cols = N / grid_rows;

//     dim3 dimGrid(grid_cols, grid_cols, 1);
//     dim3 dimBlock(grid_rows, grid_rows, 1);

//     matMul<<<dimGrid, dimBlock>>>(A, B, C, N, N, N);
//     hipDeviceSynchronize();

//     // check for errors (all vals should be 96.0f)
//     float maxError = 0.0f;
//     for (int i = 0; i < N; i++) {
//         for (int j = 0; j < N; j++)
//             maxError = fmax(maxError, fabs(C[i * N + j] - 96.0f));
//     }
//     std::cout << "Max Error: " << maxError << std::endl;

//     hipFree(A);
//     hipFree(B);
//     hipFree(C);

//     return 0;
// }